#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

// This function will execute in GPU device. A.K.A. Graphics Card
__global__ void hello_cuda()
{
	printf("Hello CUDA world \n");
}

// This function will execute in CPU Host. A.K.A. The Computer
int main()
{
  // Threads are organized in 3D Cartesian coordinates
  // Each block has a number of threads for x, y and z axes
  // The grid has a number of blocks for x, y and z axes
  // Imagine a big cube composed of smaller cubes if x=y=z
  // Or some other shape for x!=y!=z

	int threads_count_in_x_axis;
	threads_count_in_x_axis = 1; // In this example, we are using only one thread

  // number of threads per block
	dim3 block(threads_count_in_x_axis); // axes: x, y, z
  // number of blocks in grid
	dim3 grid(threads_count_in_x_axis / block.x); // axes: x, y, z

  // execute code on GPU
	hello_cuda << < grid, block >> > ();
  // wait for GPU to finish running code
	hipDeviceSynchronize();
  // Destroy all allocations and reset GPU
	hipDeviceReset();
	return 0;
}
