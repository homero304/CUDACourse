#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void hello_cuda()
{
  printf(" threadIdx.x : %d, threadIdx.y : %d, threadIdx.z : %d\n",
  threadIdx.x, threadIdx.y, threadIdx.z);
}

int main()
{
  int threads_count_in_x_axis;
  threads_count_in_x_axis = 16; 
  int threads_count_in_y_axis;
  threads_count_in_y_axis = 16;
  dim3 block(8, 8); 
  dim3 grid(threads_count_in_x_axis / block.x, threads_count_in_y_axis / block.y); 

  hello_cuda << < grid, block >> > ();
  hipDeviceSynchronize();
  hipDeviceReset();
  return 0;
}
