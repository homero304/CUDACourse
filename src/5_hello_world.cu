#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

// This function will execute in GPU device. A.K.A. Graphics Card
__global__ void hello_cuda()
{
	printf("Hello CUDA world \n");
}

// This function will execute in CPU Host. A.K.A. The Computer
int main()
{
  // Threads are organized in 3D Cartesian coordinates
  // Each block has a number of threads for x, y and z axes
  // The grid has a number of blocks for x, y and z axes
  // Imagine a big cube composed of smaller cubes if x=y=z
  // Or some other shape for x!=y!=z

  // to check thread count limitations in X Y Z for blocks and the grid,
  // compile and run ./deviceQuery from CUDA Samples
	int threads_count_in_x_axis;
	threads_count_in_x_axis = 1; // In this example, we are using only one thread
  // block limitations example -- X <= 1024, Y <= 1024, Z <= 64  --- X * Y * Z <= 1024
  // number of threads per block
	dim3 block(threads_count_in_x_axis); // axes: x, y, z
  // grid limitations example -- X <= 2147483647, Y <= 65535, Z <= 65535 -- X * Y * Z <= 2147483647
  // number of blocks in grid
	dim3 grid(threads_count_in_x_axis / block.x); // axes: x, y, z

  // execute code on GPU
	hello_cuda << < grid, block >> > ();
  // wait for GPU to finish running code
	hipDeviceSynchronize();
  // Destroy all allocations and reset GPU
	hipDeviceReset();
	return 0;
}
